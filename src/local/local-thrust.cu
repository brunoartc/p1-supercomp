#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <vector>
#include <random>
#include <tuple> // for tuple
#include <array>     // std::array
#include <iostream>  // std::cout
#include <algorithm> // std::shuffle
#include <chrono>    // std::chrono::system_clock
#include <thrust/tabulate.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <thrust/random.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/extrema.h>



/*
struct shuffle
{
    int n_alunos, total;
    int* ptr;
    thrust::uniform_int_distribution<int> dist;

    __host__ __device__
    shuffle(int _n_alunos, int _total, int* _ptr,  thrust::uniform_int_distribution<int> _dist) : n_alunos(_n_alunos), total(_total), ptr(_ptr), dist(_dist){};

    __host__ __device__
        thrust::uniform_int_distribution<int> dist;
        int operator()(const unsigned int N) const
        {
            thrust::default_random_engine rng(N);
                

            int random;
            random = dist(rng);
            //rng.discard(N);
            const int index = (((int) N/n_alunos) * n_alunos);


            int temp = ptr[index];
            //ptr[index] = ptr[N];
            //ptr[N] = 2;



            return index;//return temp;//return ptr[dist(rng)/n_alunos];//return dist(rng);//ptr[dist(rng)/n_alunos];
            
            
        }
};

 //old gold*/


 struct shuffle{
    int n_alunos;
    int total;
    int *ptr;
    int *prefs;
    
    int seed;
    thrust::uniform_int_distribution<int> dist;
    
    shuffle ( int _n_alunos, int _total, int *_ptr,
                    thrust::uniform_int_distribution<int> dist, int *_prefs) :
                                                                  n_alunos(_n_alunos),
                                                                  total(_total),
                                                                  ptr(_ptr),
                                                                  dist(dist),
                                                                  prefs(_prefs){};

    __device__ __host__
    int operator()(const int &i) {
        thrust::default_random_engine rgn(i);
        /*rgn.discard(i);
        int randommm;
        randommm = dist(rgn);
        const int index = ( i * n_alunos) + randommm;// * n_alunos) + randommm;
        int old = ptr[index];
        ptr[index] = ptr[i];
        ptr[i] = old;*/   //old


        for(int j=0; j < n_alunos; j++){
            int randommm = dist(rgn);
            int old = ptr[j+i*n_alunos];
            ptr[j+i*n_alunos] = ptr[randommm+i*n_alunos]; //faz trocas no aluno projeto aleatoriamente
            ptr[randommm+i*n_alunos] = old;
        }
        int ful_sat = 0;
        for(int j=0; j < n_alunos; j++){
            ful_sat += prefs[j+i*n_alunos + ptr[j+i*n_alunos]];
        }

        return ful_sat;
    }
};





struct make_init : public thrust::unary_function<int,int>
{
  int _a;
  make_init(int a) : _a(a) {}
  int operator()(int i) const { return i % this->_a; }
};




int main(int argc, char const *argv[])
{

    int n_alunos, n_projetos, n_choices;
    

    std::cin >> n_alunos;
    std::cin >> n_projetos;
    std::cin >> n_choices;

    thrust::host_vector<int> prefs(n_alunos*n_choices);

    for (int i = 0; i < n_alunos; i++)
    {

        std::vector<int> prefs_choice;
        int choice;
        for (int i = 0; i < n_choices; i++)
        {
            std::cin >> choice;
            prefs_choice.push_back(choice);
        }
        

        for (int j = 0; j < n_choices; j++)
        {
            prefs[i + prefs_choice[j]] = pow(n_choices - j, 2);
        }
    }
    
    thrust::device_vector<int> dprefs = prefs;

    std::vector<int> vagas(n_projetos, n_choices); // start with value of n_choices
    const int inters = 200000;
    const int N = n_alunos * inters;
    thrust::host_vector<int> A(N);   

    thrust::host_vector<int> santax(N);
    thrust::host_vector<int> santax2(N);

    thrust::device_vector<int> satis(N);

    
    thrust::tabulate(A.begin(), A.end(), make_init(n_projetos));


    thrust::device_vector<int> d_vec = A;
    thrust::device_vector<int> st = A;
    int initila = 0;
    /*for (int i = initila; i < initila +9; i++)
    {
        std::cout << A[i] << "\n";
    }*/


    thrust::counting_iterator<unsigned int> index_sequence_begin(0);
    

    int* dv_ptr = thrust::raw_pointer_cast(d_vec.data());
    int* dv_prefs_ptr = thrust::raw_pointer_cast(d_vec.data());


    thrust::uniform_int_distribution<int> dist(0,n_projetos - 1);


    thrust::transform(index_sequence_begin,
            index_sequence_begin + inters,
            satis.begin(),
            shuffle(n_alunos, N, dv_ptr, dist, dv_prefs_ptr));


    

    

    thrust::copy(satis.begin(), satis.end(), santax.begin());

    thrust::copy(d_vec.begin(), d_vec.end(), santax2.begin());
    
    int* vec = santax.data();

    int indexx  = std::distance(vec, std::max_element(vec, vec + N));


    std::cout << vec[indexx] << " 1" << "\n";
    for (int i = 0; i < n_alunos-1; i++)
    {
        std::cout << santax2[indexx*n_alunos + i] << " ";
    }
    std::cout << santax2[indexx*n_alunos + n_alunos];

    /*for (int i = initila; i < initila +9; i++)
    {
        std::cout << "i:" << i<<" <-> " << santax[i] << " " << max << "\n";
    }*/

    
}