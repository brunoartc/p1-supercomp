#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <vector>
#include <random>
#include <tuple> // for tuple
#include <array>     // std::array
#include <iostream>  // std::cout
#include <algorithm> // std::shuffle
#include <chrono>    // std::chrono::system_clock
#include <thrust/tabulate.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <thrust/random.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/iterator/counting_iterator.h>


struct shuffle
{
    int n_alunos, total;
    

    __host__ __device__
    shuffle(int _n_alunos, int _total) : n_alunos(_n_alunos), total(_total) {};

    __host__ __device__
        int operator()(const unsigned int *N) const
        {
            for (int i = 0; i < total/n_alunos; i++)
            {
                thrust::minstd_rand rng;
                
                for (int j = 0; i < n_alunos; i++)
                {
                    thrust::uniform_int_distribution<int> dist(0, 20);
                    rng.discard(i);
                    ptr[i*j] = ptr[i*dist(i)];
                }
                
                


            }
            
            
        }
};





struct make_init : public thrust::unary_function<int,int>
{
  int _a;
  make_init(int a) : _a(a) {}
  int operator()(int i) const { return i % this->_a; }
};


int main(int argc, char const *argv[])
{

    int n_alunos, n_projetos, n_choices;
    

    std::cin >> n_alunos;
    std::cin >> n_projetos;
    std::cin >> n_choices;

    std::vector<std::vector<int>> prefs(n_alunos);
    for (int i = 0; i < n_alunos; i++)
        prefs[i].resize(n_projetos);

    for (int i = 0; i < n_alunos; i++)
    {

        std::vector<int> prefs_choice;
        int choice;
        for (int i = 0; i < n_choices; i++)
        {
            std::cin >> choice;
            prefs_choice.push_back(choice);
        }
        

        for (int j = 0; j < n_choices; j++)
        {
            prefs[i][prefs_choice[j]] = pow(n_choices - j, 2);
        }
    }

    std::vector<int> vagas(n_projetos, n_choices); // start with value of n_choices
    const int N = n_alunos * 200000;
    thrust::host_vector<int> A(N);   

    
    thrust::tabulate(A.begin(), A.end(), make_init(n_projetos));


    thrust::device_vector<int> d_vec = A;

    for (int i = 0; i < 10; i++)
    {
        std::cout << A[i] << "\n";
    }


    thrust::counting_iterator<unsigned int> index_sequence_begin(0);
    

    /*thrust::transform(index_sequence_begin,
            index_sequence_begin + N,
            d_vec.begin(),
            shuffle(n_alunos, N));*/
    
    
    

    
}